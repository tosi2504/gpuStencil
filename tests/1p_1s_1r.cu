#include "hip/hip_runtime.h"
#include "error.h"
#include "gpuStencil.cuh"
#include "errorcheck.h"
#include "datatypes.cuh"

#include <iostream>

constexpr unsigned N = 64;
constexpr unsigned nRhs = 1;

int main () {
    complexF *u_A, *u_X, *u_Y;
    complexF **u_u_A, **u_u_X, **u_u_Y;
    complexF ***u_u_u_A;
    
    CCE(    hipMallocManaged(&u_A, sizeof(complexF)*N*N)     );
    CCE(    hipMallocManaged(&u_X, sizeof(complexF)*N)       );
    CCE(    hipMallocManaged(&u_Y, sizeof(complexF)*N)       );
    CCE(    hipMallocManaged(&u_u_A, sizeof(complexF*))      );
    CCE(    hipMallocManaged(&u_u_X, sizeof(complexF*))      );
    CCE(    hipMallocManaged(&u_u_Y, sizeof(complexF*))      );
    CCE(    hipMallocManaged(&u_u_u_A, sizeof(complexF**))   );
    
    *u_u_A = u_A;
    *u_u_X = u_X;
    *u_u_Y = u_Y;

    *u_u_u_A = u_u_A;
    
    for (unsigned i = 0; i < N*N; i++) {
        if (i/N == i%N) {
            u_A[i] = 1;
        } else {
            u_A[i] = 0;
        }
    }
    
    for (unsigned i = 0; i < N; i++) {
        u_X[i] = i;
    }
    
    ret_status_t res = interface1<N, nRhs, N/2, N/2, 2, 2>(u_u_Y, u_u_u_A, u_u_X, 1, 1);
    if (res != OK) {
        std::cout << "Bad return value" << std::endl;
        return 1;
    }
    
    for (unsigned i = 0; i < N; i++) {
        if (std::abs(u_X[i]-u_Y[i]) > 0.00001) {
            std::cout << "Bad diff" << std::endl;
            return 2;
        }
    }
    
    return 0;
}
